#include "hip/hip_runtime.h"
#include "cudaStepper.cuh"
#include <stdio.h>
__global__ void stepper(
	float* d_firingRate,
	float* d_newFiringRate,
	float* d_connMatrix,
	int* d_sampleNeuronIndexes,
	float* d_biasVec,
	float* d_samples,
	float* stepSize,
	int* numNeurons) {

	int neurNum = blockIdx.x;
	float fireSum = 0;

	int index;
	for (int i = 0; i < (*numNeurons); i++) {
		index = neurNum * (*numNeurons) + i;
		fireSum += d_firingRate[i] * d_connMatrix[index];
	}
	fireSum += d_biasVec[neurNum];
	float fVal = 1 / (1 + exp(-fireSum));
	fVal -= d_firingRate[neurNum];
	d_newFiringRate[neurNum] = d_firingRate[neurNum] + (fVal * (*stepSize));
	index = d_sampleNeuronIndexes[neurNum];
	if (index > -1) {
		//printf("firingRate sample: %f\n", d_newFiringRate[neurNum]);
		d_samples[index] = d_newFiringRate[neurNum];
	}

}


namespace NNet {
	float** stepSys(
		int numSteps,
		int numNeurons,
		float** connMatrix,
		float* biasVec,
		float* startRate,
		std::vector<int> sampleNeurons,
		float stepSize
	) {
		//Set up flag array for sampled neurons
		int* d_sampleNeuronIndexes;
		int* sampleNeuronFlags = new int[numNeurons];
		for (int i = 0; i < numNeurons; i++) {
			sampleNeuronFlags[i] = -1;
		}
		float* sampleLayer = new float[sampleNeurons.size()];
		float** sampleRates = new float* [numSteps + 1];
		int acc = 0;
		for (int i = 0; i < sampleNeurons.size(); i++) {
			sampleNeuronFlags[sampleNeurons[i]] = acc;
			acc++;
			sampleLayer[i] = startRate[sampleNeurons[i]];
		}
		sampleRates[0] = sampleLayer;
		int size = sizeof(int) * numNeurons;
		hipMalloc((void**)&d_sampleNeuronIndexes, size);
		hipMemcpy(d_sampleNeuronIndexes, sampleNeuronFlags, size, hipMemcpyHostToDevice);

		float* d_samples;
		size = sampleNeurons.size() * sizeof(float);
		hipMalloc((void**)&d_samples, size);
		
		//Copying to GPU system properties
		float* d_biasVec; float* d_firingRate;
		float* connMatrix1D; float* d_connMatrix;
		float* d_newFiringRate;

		int index;
		connMatrix1D = new float[numNeurons * numNeurons];
		for (int i = 0; i < numNeurons; i++) {
			for (int j = 0; j < numNeurons; j++) {
				index = i * numNeurons + j;
				connMatrix1D[index] = connMatrix[i][j];
			}
		}
		size = sizeof(float) * numNeurons * numNeurons;
		hipMalloc((void**)&d_connMatrix, size);
		hipMemcpy(d_connMatrix, connMatrix1D, size, hipMemcpyHostToDevice);

		size = sizeof(float) * numNeurons;
		hipMalloc((void**)&d_biasVec, size); hipMalloc((void**)&d_firingRate, size);
		hipMemcpy(d_biasVec, biasVec, size, hipMemcpyHostToDevice);
		hipMemcpy(d_firingRate, startRate, size, hipMemcpyHostToDevice);
		hipMalloc((void**)&d_newFiringRate, size);

		size = sizeof(int);
		int* d_numNeur; hipMalloc((void**)&d_numNeur, size);
		hipMemcpy(d_numNeur, &numNeurons, size, hipMemcpyHostToDevice);

		size = sizeof(float);
		float* d_stepSize; hipMalloc((void**)&d_stepSize, size);
		hipMemcpy(d_stepSize, &stepSize, size, hipMemcpyHostToDevice);

		int sizeSample = sizeof(float) * sampleNeurons.size();
		int sizeUpdate = sizeof(float) * numNeurons;
		
		

		//Stepping system over input number of steps using stepper cuda kernel.
		/*for (int i = 0; i < numSteps; i++) {
			stepper<<<numNeurons, 1>>>(d_firingRate, d_newFiringRate, d_connMatrix, d_sampleNeuronIndexes,
				d_biasVec, d_samples, d_stepSize, d_numNeur);
			float* sampleLayer = new float[sampleNeurons.size()];
			hipMemcpy(sampleLayer, d_samples, sizeSample, hipMemcpyDeviceToHost);
			sampleRates[i+1] = sampleLayer;
			hipMemcpy(d_firingRate, d_newFiringRate, sizeUpdate, hipMemcpyDeviceToDevice);
		}*/

		//stepping system over input number of steps... Sin(t) bias'
		float time = 0.0;
		int sizeBias = sizeof(float) * numNeurons;
		
		for (int i = 0; i < numSteps; i++) {
			float sint = sin(time);
			for (int i = 0; i < numNeurons; i++) {
				biasVec[i] = abs(sint);
			}
			time += stepSize;
			hipMemcpy(d_biasVec, biasVec, sizeBias, hipMemcpyHostToDevice);
			stepper << <numNeurons, 1 >> > (d_firingRate, d_newFiringRate, d_connMatrix, d_sampleNeuronIndexes,
				d_biasVec, d_samples, d_stepSize, d_numNeur);
			float* sampleLayer = new float[sampleNeurons.size()];
			hipMemcpy(sampleLayer, d_samples, sizeSample, hipMemcpyDeviceToHost);
			sampleRates[i+1] = sampleLayer;
			hipMemcpy(d_firingRate, d_newFiringRate, sizeUpdate, hipMemcpyDeviceToDevice);
		}
		
		return sampleRates;
	}
}


